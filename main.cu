#include <iostream>
#include <time.h>

#include "includes/cuda-fuzzy/lib/FuzzyLogic/FuzzyLogic.cu"
#include "includes/cuda-fuzzy/lib/FuzzyLogic/BulkLogic.cu"
#include "includes/cuda-fuzzy/lib/FuzzyLogic/d_BulkLogic.cu"

using namespace std;

double* generateRandomArray(int size);
double fRand(double fMin, double fMax);

int main() {

	float deviceTime, hostTime;

	int arraySize;

	cout << "arraySize,operation,sequentialTime,cudaTime" << endl;

	for(arraySize = 10000; arraySize < 1000000000; arraySize = arraySize*10) {

		for(int i = 0; i < 30; i++) {

			double* array1 = generateRandomArray(arraySize);
			double* array2 = generateRandomArray(arraySize);
			double* result;

			hipEvent_t start, stop;

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = h_BulkNot(array1, arraySize);
			hipEventRecord(stop, 0);

			free(result);
			
			hipEventSynchronize(stop);

			hipEventElapsedTime(&hostTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = d_BulkNot(array1, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&deviceTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			cout << arraySize << ",Not," << hostTime << "," << deviceTime << endl;

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = h_BulkNot2(array1, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&hostTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = d_BulkNot2(array1, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&deviceTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			cout << arraySize << ",Not2," << hostTime << "," << deviceTime << endl;

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = h_BulkNot3(array1, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&hostTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = d_BulkNot3(array1, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&deviceTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			cout << arraySize << ",Not3," << hostTime << "," << deviceTime << endl;

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = h_BulkAnd(array1, array2, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&hostTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = d_BulkAnd(array1, array2, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&deviceTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			cout << arraySize << ",And," << hostTime << "," << deviceTime << endl;

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = h_BulkAnd2(array1, array2, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&hostTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = d_BulkAnd2(array1, array2, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&deviceTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			cout << arraySize << ",And2," << hostTime << "," << deviceTime << endl;

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = h_BulkOr(array1, array2, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&hostTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = d_BulkOr(array1, array2, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&deviceTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			cout << arraySize << ",Or," << hostTime << "," << deviceTime << endl;

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = h_BulkOr2(array1, array2, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&hostTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			result = d_BulkOr2(array1, array2, arraySize);
			hipEventRecord(stop, 0);

			free(result);

			hipEventSynchronize(stop);

			hipEventElapsedTime(&deviceTime, start, stop);
			
			hipEventDestroy(start);
			hipEventDestroy(stop);

			cout << arraySize << ",Or2," << hostTime << "," << deviceTime << endl;

			free(array1);
			free(array2);
		}
	}	

	return 1;

}

double* generateRandomArray(int size) {

	double* array = (double*)malloc(sizeof(double) * size);
	srand(time(NULL));

	for(int i = 0; i < size; i++) {

		array[0] = fRand(0, 1);

	}

	return array;

}

double fRand(double fMin, double fMax) {
	double f = (double)rand() / RAND_MAX;
 	return fMin + f * (fMax - fMin);
}